#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <cmath>
#include <math.h>
#include <cstdio>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

#define FILTER_RADIUS 2
const float F_h[2 * FILTER_RADIUS + 1][2 * FILTER_RADIUS + 1] = {
    {1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25},
    {1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25},
    {1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25},
    {1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25},
    {1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25, 1.0f / 25}};
__constant__ float F[2 * FILTER_RADIUS + 1][2 * FILTER_RADIUS + 1];

/**
    module load opencv/4.9.0.x86_64
*/

/******************************************************************************************************* */
/* Helper Functions*/
/* START */

#define CHECK(call)                                                                  \
    {                                                                                \
        const hipError_t cuda_ret = call;                                           \
        if (cuda_ret != hipSuccess)                                                 \
        {                                                                            \
            printf("Error: %s:%d, ", __FILE__, __LINE__);                            \
            printf("code: %d, reason:%s\n", cuda_ret, hipGetErrorString(cuda_ret)); \
            exit(-1);                                                                \
        }                                                                            \
    }

double myCPUTimer()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec / 1.0e6);
}

bool verify(cv::Mat answer1, cv::Mat answer2, unsigned int nRows, unsigned int nCols)
{
    const float relativeTolerance = 1e-2;

    for (int i = 0; i < nRows; i++)
    {
        for (int j = 0; j < nCols; j++)
        {
            float relativeError = ((float)answer1.at<unsigned char>(i, j) - (float)answer2.at<unsigned char>(i, j)) / 255;
            if (relativeError > relativeTolerance || relativeError < -relativeTolerance)
            {
                printf("TEST FAILED at (%d, %d) with relativeError: %f\n", i, j, relativeError);
                printf("    answer1.at<unsigned char>(%d, %d): %u\n", i, j, answer1.at<unsigned char>(i, j));
                printf("    answer2.at<unsigned char>(%d, %d): %u\n\n", i, j, answer2.at<unsigned char>(i, j));
                return false;
            }
        }
    }

    printf("TEST PASSED\n\n");
    return true;
}

/* END */
/* Helper Functions*/
/******************************************************************************************************* */

/******************************************************************************************************* */
/* Convolution Functions*/
/* START */

Mat opencv_convolution(Mat bwImage)
{
    Mat kernel1 = Mat::ones(5, 5, CV_64F);
    kernel1 = kernel1 / 25;
    Mat blurred;
    filter2D(bwImage, blurred, -1, kernel1);
    return blurred;
}

void blurImage_h(Mat &Pout_Mat_h, const cv::Mat &Pin_Mat_h, unsigned int nRows, unsigned int nCols)
{

    Pout_Mat_h = Mat::zeros(nRows, nCols, CV_8U);

    const float filterValue = 1.0f / 25.0f;

    for (int i = FILTER_RADIUS; i < nRows - FILTER_RADIUS; i++)
    {
        for (int j = FILTER_RADIUS; j < nCols - FILTER_RADIUS; j++)
        {

            float sum = 0.0f;
            for (int k = -FILTER_RADIUS; k <= FILTER_RADIUS; k++)
            {
                for (int l = -FILTER_RADIUS; l <= FILTER_RADIUS; l++)
                {
                    sum += filterValue * Pin_Mat_h.at<unsigned char>(i + k, j + l);
                }
            }
            Pout_Mat_h.at<unsigned char>(i, j) = sum;
        }
    }
}

__global__ void blurImage_Kernel(unsigned char *Pout, unsigned char *Pin, unsigned int width, unsigned int height)
{
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;

    if (outCol < width && outRow < height)
    {
        float Pvalue = 0.0f;
        int inRow = 0, inCol = 0;
        int condition = 2 * FILTER_RADIUS + 1;

        for (int fRow = 0; fRow < condition; fRow++)
        {
            for (int fCol = 0; fCol < condition; fCol++)
            {
                inRow = outRow - FILTER_RADIUS + fRow;
                inCol = outCol - FILTER_RADIUS + fCol;
                if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width)
                {
                    Pvalue += F[fRow][fCol] * (float)Pin[inRow * width + inCol];
                }
            }
        }

        Pout[outRow * width + outCol] = (unsigned char)min(max(Pvalue, 0.0f), 255.0f);
    }
}

void blurImage_d(cv::Mat Pout_Mat_h, cv::Mat Pin_Mat_h, unsigned int nRows, unsigned int nCols)
{

    printf("\n\nblurImage_Kernel: \n");

    // (1) allocate device memory for arrays p_d
    unsigned char *Pin_d, *Pout_d;
    double start_time_malloc = myCPUTimer();
    hipMalloc((void **)&Pin_d, sizeof(unsigned char) * nRows * nCols);
    hipMalloc((void **)&Pout_d, sizeof(unsigned char) * nRows * nCols);
    double end_time_malloc = myCPUTimer();
    double elapsed_time_malloc = end_time_malloc - start_time_malloc;

    printf("\tcudaMalloc: \t\t\t\t\t\t\t\t%f s\n", elapsed_time_malloc);

    // (2) copy image matrix Pin_h to device memory Pin_d
    unsigned char *Pin_h = Pin_Mat_h.data;
    double start_time_memcpy = myCPUTimer();
    hipMemcpy(Pin_d, Pin_h, sizeof(unsigned char) * nCols * nRows, hipMemcpyHostToDevice);
    double end_time_memcpy = myCPUTimer();
    double elapsed_time_memcpy = end_time_memcpy - start_time_memcpy;

    printf("\tcudaMemcpy: \t\t\t\t\t\t\t\t%f s\n", elapsed_time_memcpy);

    // (3) call kernel to launch a grid of threads to perform the image convolution on GPU
    dim3 gridDim((nRows + 32 - 1) / 32, (nCols + 32 - 1) / 32);
    dim3 blockDim(32, 32);

    double start_time = myCPUTimer();
    blurImage_Kernel<<<gridDim, blockDim>>>(Pout_d, Pin_d, nRows, nCols);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    double end_time = myCPUTimer();
    double elapsed_time = end_time - start_time;

    printf("\tblurImage_Kernel<<<(%d, %d, 1), (%d, %d, 1)>>>: \t\t\t%f s\n", (nCols + 32 - 1) / 32, (nRows + 32 - 1) / 32, 32, 32, elapsed_time);

    // (4) Copy the result data from device memory of array Pout_d to host memory of array Pout_h
    Pout_Mat_h = cv::Mat::zeros(nRows, nCols, CV_8U);
    unsigned char *Pout_h = Pout_Mat_h.data;
    double start_time_memcpy2 = myCPUTimer();
    hipMemcpy(Pout_h, Pout_d, sizeof(unsigned char) * nCols * nRows, hipMemcpyDeviceToHost);
    double end_time_memcpy2 = myCPUTimer();
    double elapsed_time_memcpy2 = end_time_memcpy2 - start_time_memcpy2;

    // printf("\n\n\n");
    // for(int i = 0; i < nCols*nRows; i++){
    //     printf("%hhu ", Pout_h);
    // }
    // printf("\n\n\n");

    printf("\tcudaMemcpy: \t\t\t\t\t\t\t\t%f s\n\n", elapsed_time_memcpy2);

    double total_elapsed_time = elapsed_time_malloc + elapsed_time_memcpy + elapsed_time + elapsed_time_memcpy2;

    printf("Total elapsed time for convolution without tiling: %f s\n", total_elapsed_time);

    // (5) free device memory of Pin_d and Pout_d
    hipFree(Pin_d);
    hipFree(Pout_d);
}

/* Convolution Functions*/
/* END */
/******************************************************************************************************* */

int main(int argc, char *argv[])
{

    char *file_name = argv[1];
    printf("Blurring file: \"%s\"\n", file_name);

    // for comparison purpose, use OpenCV's 2D Filter function
    Mat Pin_Mat_h = cv::imread(file_name, IMREAD_GRAYSCALE);
    unsigned int nRows = Pin_Mat_h.rows, nCols = Pin_Mat_h.cols, nChannels = Pin_Mat_h.channels();

    printf("\n\n\n Dimension of image: %d %d \n\n\n", nRows, nCols);

    double start_time = myCPUTimer();
    Mat blurred_mat = opencv_convolution(Pin_Mat_h);
    double end_time = myCPUTimer();
    double elapsed_time = end_time - start_time;

    printf("\nOpenCV filter2D(image, blurred,  -1, kernel1): %f s\n", elapsed_time);

    // for comparison purpose, implement a CPU version
    Mat blurred_imaged;
    start_time = myCPUTimer();
    blurImage_h(blurred_imaged, Pin_Mat_h, nRows, nCols);
    end_time = myCPUTimer();
    elapsed_time = end_time - start_time;

    std::time_t t = std::time(nullptr);
    std::tm *now = std::localtime(&t);

    std::ostringstream oss;
    oss << "blurred_("
        << (now->tm_year + 1900) << '-'
        << (now->tm_mon + 1) << '-'
        << now->tm_mday << '_'
        << now->tm_sec
        << ").jpg";

    std::string filename = oss.str();
    imwrite(filename, blurred_imaged);
    printf("\nCPU Version blurImage_h(blurred_imaged, image, nRows, nCols): %f s\n", elapsed_time);

    // for comparison purpose, implement a CUDA kernel but without tiling
    Mat Pout_Mat_h(nRows, nCols, CV_8U);
    hipMemcpyToSymbol(HIP_SYMBOL(F), F_h, (2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1) * sizeof(float));
    blurImage_d(Pout_Mat_h, Pin_Mat_h, nRows, nCols);

    std::ostringstream oss_kernel;
    oss_kernel << "blurred_kernel("
               << (now->tm_year + 1900) << '-'
               << (now->tm_mon + 1) << '-'
               << now->tm_mday << '_'
               << now->tm_sec
               << ").jpg";

    std::string filename_kernel = oss_kernel.str();
    imwrite(filename_kernel, Pout_Mat_h);

    // for comparison purpose, implement a CUDA kernel but with tiling and constant memory

    return 0;
}
