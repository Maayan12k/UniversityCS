
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

/**
  *  Must use cudaDeviceSynchronize() when measuring GPU kernel operations because they are non blocking. 
 */
double myCPUTimer(){ 
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec/1.0e6);
}

void vecAdd_h(float* x_h, float* y_h, float* z_h, unsigned int n){
    for(unsigned int i = 0; i < n; i++)
        z_h[i] = x_h[i] + y_h[i];
}

__global__ void vecAddKernel(float* x_d, float* y_d, float* z_d, unsigned int n) {
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < n) // handling boundary conditions
        z_d[i] = x_d[i] + y_d[i];
}

int main(int argc, char** argv){

    unsigned int n = 1024;

    //allocate host memory for x_h, y_h, and z_h and intialize x_h, y_h
    float* x_h = (float*) malloc(sizeof(float)*n);
    for(unsigned int i = 0; i < n; i++) x_h[i] = (float) rand()/(float)(RAND_MAX);
    float* y_h = (float*) malloc(sizeof(float)*n);
    for(unsigned int i = 0; i < n; i++) y_h[i] = (float) rand()/(float)(RAND_MAX);
    float* z_h = (float*) calloc(n, sizeof(float));

    // (1) allocate device memory for arrays x_d, y_d, z_d
    float *x_d, *y_d, *z_d;
    hipMalloc((void**) &x_d, sizeof(float)*n);
    hipMalloc((void**) &y_d, sizeof(float)*n);
    hipMalloc((void**) &z_d, sizeof(float)*n);

    // (2) copy arrays x_h and y_h to device memory x_d and y_d, respectively
    hipMemcpy(x_d, x_h, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(float)*n, hipMemcpyHostToDevice);

    // (3) call kernel to launch a grid of threads to perform the vector addition on GPU && CPU
    double startTime_h = myCPUTimer();
    vecAdd_h(x_h, y_h, z_h, n);
    double endTime_h = myCPUTimer();
    printf("Execution time of CPU 1024 Vector addition: %f\n", endTime_h - startTime_h);

    double startTime_d = myCPUTimer();
    vecAddKernel<<<ceil(n/256.0), 256>>>(x_d, y_d, z_d, n);
    hipDeviceSynchronize();
    double endTime_d = myCPUTimer();
    printf("Execution time of GPU 1024 Vector addition: %f\n", endTime_d - startTime_d);

    // (4) Copy the result data from device memory of array  z_d to host memory of array z_h
    hipMemcpy(z_h, z_d, sizeof(float)*n, hipMemcpyDeviceToHost);

    // (5) free device memory of x_d, y_d, and z_d 
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);

    // free host memory of x_h, z_h, and z_h
    free(x_h);
    free(y_h);
    free(z_h);

    return 0;
}
