#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <cmath>
#include <math.h>

/******************************************************************************************************* */
/* Helper Functions*/
/* START */

// Must use hipDeviceSynchronize() when measuring GPU kernel operations because CUDA kernel operations are non blocking.
double myCPUTimer()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec / 1.0e6);
}

bool verify(float *gpu_result, float *cpu_result, unsigned int nRows, unsigned int nCols, int precision)
{
    const float epsilon = std::pow(10, -precision);

    for (int i = 0; i < nRows * nCols; i++)
    {

        if (std::fabs(cpu_result[i] - gpu_result[i]) > epsilon)
        {
            return false;
        }
    }
    return true;
}

int calculatePrecision(int m, int n, int k)
{
    int totalOperations = m * n * k;
    const int C = 15;
    int precision = (int)fmax(1, C / log10(totalOperations));
    return precision;
}

/* END */
/* Helper Functions*/
/******************************************************************************************************* */

/******************************************************************************************************* */
/* Matrix Multiplication Functions*/
/* START */
void basicSgemm_h(float *a_h, float *b_h, float *c_h, unsigned int m, unsigned int k, unsigned int n)
{

    for (int outputMatrixIndex = 0; outputMatrixIndex < m * n; outputMatrixIndex++)
    {
        int row = outputMatrixIndex / n;
        int col = outputMatrixIndex % n;

        float sum = 0.0;

        for (int i = 0; i < k; i++)
            sum += a_h[row * k + i] * b_h[i * n + col];

        c_h[outputMatrixIndex] = sum;
    }
}

__global__ void matrixMulKernel_1thread1element(float *a_d, float *b_d, float *c_d, unsigned int m, unsigned int k, unsigned int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (col < n && row < m)
    {
        for (int i = 0; i < k; i++)
        {
            sum += a_d[row * k + i] * b_d[i * n + col];
        }
        c_d[row * n + col] = sum;
    }
}

__global__ void matrixMulKernel_1thread1row(float *a_d, float *b_d, float *c_d, unsigned int m, unsigned int k, unsigned int n)
{

    int row = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global row index
    if (row < m)
    { // Check if the row index is within bounds
        for (int col = 0; col < n; col++)
        {
            float sum = 0.0;
            for (int i = 0; i < k; i++)
            {
                sum += a_d[row * k + i] * b_d[i * n + col];
            }
            c_d[row * n + col] = sum;
        }
    }
}

__global__ void matrixMulKernel_1thread1column(float *a_d, float *b_d, float *c_d, unsigned int m, unsigned int k, unsigned int n)
{

    int col = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global column index
    if (col < n)
    { // Check if the column index is within bounds
        for (int row = 0; row < m; row++)
        {
            float sum = 0.0;
            for (int i = 0; i < k; i++)
            {
                sum += a_d[row * k + i] * b_d[i * n + col];
            }
            c_d[row * n + col] = sum;
        }
    }
}
/* Matrix Multiplication Functions*/
/* END */
/******************************************************************************************************* */

void basicSgemm_d_1thread1element(float *a_h, float *b_h, float *c_h, unsigned int m, unsigned int k, unsigned int n)
{

    // (1) allocate device memory for arrays x_d, y_d, z_d
    float *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, sizeof(float) * m * k);
    hipMalloc((void **)&b_d, sizeof(float) * k * n);
    hipMalloc((void **)&c_d, sizeof(float) * m * n);

    // (2) copy matrices a_h and b_h to device memory a_d and b_d, respectively
    hipMemcpy(a_d, a_h, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, sizeof(float) * k * n, hipMemcpyHostToDevice);

    // (3) call kernel to launch a grid of threads to perform the matrix multiplcation on GPU
    dim3 gridDim((n + 16 - 1) / 16, (m + 16 - 1) / 16);
    dim3 blockDim(16, 16);

    double start_time = myCPUTimer();
    matrixMulKernel_1thread1element<<<gridDim, blockDim>>>(a_d, b_d, c_d, m, k, n);
    hipDeviceSynchronize();
    double end_time = myCPUTimer();
    double elapsed_time = end_time - start_time;

    printf("\nElapsed time of 1 thread 1 output element: %f s\n", elapsed_time);

    // (4) Copy the result data from device memory of array c_d to host memory of array c_h
    hipMemcpy(c_h, c_d, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    // (5) free device memory of a_d, b_d, and c_d
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

void basicSgemm_d_1thread1row(float *a_h, float *b_h, float *c_h, unsigned int m, unsigned int k, unsigned int n)
{

    // (1) allocate device memory for arrays x_d, y_d, z_d
    float *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, sizeof(float) * m * k);
    hipMalloc((void **)&b_d, sizeof(float) * k * n);
    hipMalloc((void **)&c_d, sizeof(float) * m * n);

    // (2) copy matrices a_h and b_h to device memory a_d and b_d, respectively
    hipMemcpy(a_d, a_h, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, sizeof(float) * k * n, hipMemcpyHostToDevice);

    // (3) call kernel to launch a grid of threads to perform the matrix multiplcation on GPU && CPU
    dim3 gridDim;
    dim3 blockDim;
    if (m <= 1024)
    {
        gridDim.x = 1;
        blockDim.x = m;
    }
    else
    {
        gridDim.x = (m + 1023) / 1024;
        blockDim.x = 1024;
    }

    double start_time = myCPUTimer();
    matrixMulKernel_1thread1row<<<gridDim, blockDim>>>(a_d, b_d, c_d, m, k, n);
    hipDeviceSynchronize();
    double end_time = myCPUTimer();
    double elapsed_time = end_time - start_time;

    printf("Elapsed time of 1 thread 1 output row: %f s\n", elapsed_time);

    // (4) Copy the result data from device memory of array c_d to host memory of array c_h
    hipMemcpy(c_h, c_d, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    // (5) free device memory of a_d, b_d, and c_d
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

void basicSgemm_d_1thread1column(float *a_h, float *b_h, float *c_h, unsigned int m, unsigned int k, unsigned int n)
{

    // (1) allocate device memory for arrays x_d, y_d, z_d
    float *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, sizeof(float) * m * k);
    hipMalloc((void **)&b_d, sizeof(float) * k * n);
    hipMalloc((void **)&c_d, sizeof(float) * m * n);

    // (2) copy matrices a_h and b_h to device memory a_d and b_d, respectively
    hipMemcpy(a_d, a_h, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, sizeof(float) * k * n, hipMemcpyHostToDevice);

    // (3) call kernel to launch a grid of threads to perform the matrix multiplcation on GPU
    dim3 gridDim;
    dim3 blockDim;
    if (n <= 1024)
    {
        gridDim.x = 1;
        blockDim.x = n;
    }
    else
    {
        gridDim.x = (n + 1023) / 1024;
        blockDim.x = 1024;
    }

    double start_time = myCPUTimer();
    matrixMulKernel_1thread1column<<<gridDim, blockDim>>>(a_d, b_d, c_d, m, k, n);
    hipDeviceSynchronize();
    double end_time = myCPUTimer();
    double elapsed_time = end_time - start_time;

    printf("Elapsed time of 1 thread 1 output column: %f s\n", elapsed_time);

    // (4) Copy the result data from device memory of array c_d to host memory of array c_h
    hipMemcpy(c_h, c_d, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    // (5) free device memory of a_d, b_d, and c_d
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

int main(int argc, char *argv[])
{

    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);

    srand(time(0));

    // matrix 𝐴 is of size 𝑚 × 𝑘, matrix 𝐵 is of size 𝑘 × 𝑛, and matrix 𝐶 is of size 𝑚 × 𝑛.
    float *a_h = (float *)malloc(sizeof(float) * m * k);
    for (unsigned int i = 0; i < m * k; i++)
        a_h[i] = rand() % 100 / 100.0;

    float *b_h = (float *)malloc(sizeof(float) * k * n);
    for (unsigned int i = 0; i < k * n; i++)
        b_h[i] = rand() % 100 / 100.0;

    float *c_h = (float *)calloc(m * n, sizeof(float));

    float *cpu_result = (float *)calloc(m * n, sizeof(float));

    int precision = calculatePrecision(m, k, n);

    basicSgemm_h(a_h, b_h, cpu_result, m, k, n);

    printf("\nPrecision Threshold: %d decimal places.\n", precision);
    printf("\nMatrix Dimensions: \n");
    printf("\tA: %d x %d\n", m, k);
    printf("\tB: %d x %d\n", k, n);
    printf("\tC: %d x %d\n", m, n);

    bool testsPassed = true;

    basicSgemm_d_1thread1element(a_h, b_h, c_h, m, k, n);
    if (!verify(c_h, cpu_result, m, n, precision))
        testsPassed = false;

    basicSgemm_d_1thread1row(a_h, b_h, c_h, m, k, n);
    if (!verify(c_h, cpu_result, m, n, precision))
        testsPassed = false;

    basicSgemm_d_1thread1column(a_h, b_h, c_h, m, k, n);
    if (!verify(c_h, cpu_result, m, n, precision))
        testsPassed = false;

    if (testsPassed)
    {
        printf("\nVerifying Results... Tests Passed!\n");
    }
    else
    {
        printf("\nVerifying Results... Tests Failed!\n");
    }

    free(a_h);
    free(b_h);
    free(c_h);
    free(cpu_result);

    return 0;
}